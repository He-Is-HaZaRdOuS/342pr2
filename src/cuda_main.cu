#include "hip/hip_runtime.h"
/**
 *
 * CENG342 Project-2
 *
 * Edge Detection
 *
 * Usage:  executable <input.jpg> <output.jpg> <threadsPerBlock> <sequential_output.jpg>
 *
 * @group_id 07
 * @author  Yousif
 * @author  Türker
 * @author  Eren
 * @author  Aysara
 *
 * @version 1.0, 18 May 2024
 */

// ReSharper disable CppUseAuto
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <iostream>
#include "hip/hip_runtime.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#define CHANNEL_NUM 1
#define KERNEL_DIMENSION 3
#define THRESHOLD 40
#define USE_THRESHOLD 0

__constant__ float sobelX[KERNEL_DIMENSION][KERNEL_DIMENSION] = { {-1, 0, 1},{-2, 0, 2},{-1, 0, 1} };
__constant__ float sobelY[KERNEL_DIMENSION][KERNEL_DIMENSION] = { {-1, -2, -1},{0, 0, 0},{1, 2, 1} };

//Do not use global variables

/* CUDA Kernel that applies Sobel's filter */
__global__ void CUDAedgeDetection(const uint8_t* img, uint8_t* buffer, const uint64_t width, const uint64_t height) {
    /* Sobel Convolution Kernels */
    float slider[KERNEL_DIMENSION][KERNEL_DIMENSION] = {0.0};

    /* Find X and Y indices of current thread */
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    for(int wy = 0; wy < KERNEL_DIMENSION; ++wy) {
        for(int wx = 0; wx < KERNEL_DIMENSION; ++wx) {
            int xIndex = (x + wx - 1);
            int yIndex = (y + wy - 1);

            /* Clamp */
            if(xIndex <= 0)
                xIndex = -xIndex;
            if(yIndex <= 0)
                yIndex = -yIndex;
            if(xIndex >= width)
                xIndex = width - 1;
            if(yIndex >= height)
                yIndex = height - 1;

            /* Build up sliding window */
            slider[wy][wx] = img[xIndex + yIndex * width];
        }
    }

    float sumX = 0;
    float sumY = 0;

    /* Convolve */
    for(int ky = 0; ky < KERNEL_DIMENSION; ++ky) {
        for(int kx = 0; kx < KERNEL_DIMENSION; ++kx) {
            sumX = sumX + slider[kx][ky] * sobelX[kx][ky];
            sumY = sumY + slider[kx][ky] * sobelY[kx][ky];
        }
    }
    float magnitude = sqrtf((sumX*sumX)+(sumY*sumY));

#if USE_THRESHOLD
    /* Clamp down color values if below THRESHOLD */
    buffer[y * width + x] = magnitude > THRESHOLD ? 255 : 0;
#else
    /* Otherwise use whatever value outputted from square root */
    buffer[y * width + x] = static_cast<uint8_t>(magnitude);
#endif
}

class Timer {
public:
    Timer() {
        m_StartTimepoint = std::chrono::high_resolution_clock::now();
    }

    ~Timer() = default;

    double Stop() {
        m_EndTimepoint = std::chrono::high_resolution_clock::now();

        const uint64_t start = std::chrono::time_point_cast<std::chrono::microseconds>(m_StartTimepoint).time_since_epoch().count();
        const uint64_t end = std::chrono::time_point_cast<std::chrono::microseconds>(m_EndTimepoint).time_since_epoch().count();

        const std::chrono::duration<uint64_t, std::ratio<1, 1000000>>::rep duration = (end - start);
        return static_cast<double>(duration) * 0.000001;
    }
private:
    std::chrono::time_point<std::chrono::high_resolution_clock> m_StartTimepoint;
    std::chrono::time_point<std::chrono::high_resolution_clock> m_EndTimepoint;
};

int main(int argc, char* argv[]) {

    /* Abort if # of CLA is invalid */
    if(argc != 5){
        std::cerr << "Invalid number of arguments, aborting...\n";
        std::cerr << "Try ./cuda <input.jpg> <output.jpg> <threadsPerBlock> <sequential.jpg>\n";
        exit(1);
    }
    int width, height, bpp;

    /* Prepend path to input and output filenames */
    std::string inputPath = RESOURCES_PATH;
    std::string outputPath = CUDA_OUTPUT_PATH;
    inputPath = inputPath + argv[1];
    outputPath = outputPath + argv[2];
    const int threadsPerBlock = std::stoi(argv[3]);
    if(threadsPerBlock <= 0) {
        std::cerr << "Invalid argument provided, aborting...\n";
        std::cerr << "Argument <threadsPerBlock> should be a positive integer bigger than 0\n";
        exit(1);
    }
    const int threadsPerDimension = sqrt(threadsPerBlock);

    /* Read image in grayscale */
    uint8_t *input_image = stbi_load(inputPath.c_str(), &width, &height, &bpp, CHANNEL_NUM);

    if(stbi_failure_reason()) {
        std::cerr << stbi_failure_reason() << " \"" + inputPath + "\"\n";
        std::cerr << "Aborting...\n";
        exit(1);
    }

    printf("Width: %d  Height: %d  BPP: %d \n",width, height, bpp);
    printf("Input: %s , Output: %s  \n",inputPath.c_str(), outputPath.c_str());

    uint8_t *cuda_input, *cuda_output;

    /* allocate vram for buffers */
    hipMalloc(reinterpret_cast<void **>(&cuda_input), (width*height));
    hipMalloc(reinterpret_cast<void **>(&cuda_output), (width*height));

    /* memcpy input buffer from system memory into vram */
    hipMemcpy(cuda_input, input_image, width * height, hipMemcpyHostToDevice);
    /* memset output buffer inside vram */
    hipMemset(cuda_output, 0, width * height);

    /* Launch ((width*height)/GRID_SIZE^2) many SM blocks */
    dim3 blkCnt(ceil(width/threadsPerDimension), ceil(height/threadsPerDimension), 1);
    /* Assign GRID_SIZE size chunks for each SP */
    dim3 thPerBlk(threadsPerDimension, threadsPerDimension, 1);

    std::cout << "SM blocks queued up: " << blkCnt.x*blkCnt.y << ". SP threads queued up: " << thPerBlk.x*thPerBlk.y << ". Total thrads queued up: " << blkCnt.x*blkCnt.y*thPerBlk.x*thPerBlk.y << std::endl;

    /* Start the timer */
    Timer t;

    /* Call CUDA Kernel */
    CUDAedgeDetection<<<blkCnt, thPerBlk>>>(cuda_input, cuda_output, width, height);
    /* Capture errors, if any */
    hipError_t cuda_err = hipDeviceSynchronize();

    /* Stop the timer */
    double elapsedTime = t.Stop();

    /* Write error to stdout */
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "CUDA Synchronization failed!: %s\n", hipGetErrorName(cuda_err));
        exit(1);
    }

    printf("Elapsed time: %lf seconds (%lf ms) \n",elapsedTime, elapsedTime*1000);

    /* memcpy output buffer from vram back to input buffer in system memory */
    hipMemcpy(input_image, cuda_output, width * height, hipMemcpyDeviceToHost);

    /* free vram buffers */
    hipFree(cuda_input);
    hipFree(cuda_output);

    /* Write input buffer to disk */
    stbi_write_jpg(outputPath.c_str(), width, height, CHANNEL_NUM, input_image, 100);
    stbi_image_free(input_image);

    /* Check if the two image outputs are identical */
    {
        /* Prepend path to input and output filenames */
        std::string alt_input = SEQUENTIAL_OUTPUT_PATH;
        std::string par_input = CUDA_OUTPUT_PATH;
        alt_input = alt_input + argv[4];
        par_input = par_input + argv[2];
        uint8_t *alt_img, *par_img;
        int seq_width, seq_height, seq_bpp;
        int par_width, par_height, par_bpp;

        /* Read image in grayscale */
        alt_img = stbi_load(alt_input.c_str(), &seq_width, &seq_height, &seq_bpp, CHANNEL_NUM);

        /* If image could not be opened, Abort */
        if(stbi_failure_reason()) {
            std::cerr << stbi_failure_reason() << " \"" + alt_input + "\"\n";
            std::cerr << "Aborting...\n";
            exit(1);
        }

        par_img = stbi_load(par_input.c_str(), &par_width, &par_height, &par_bpp, CHANNEL_NUM);

        /* If image could not be opened, Abort */
        if(stbi_failure_reason()) {
            std::cerr << stbi_failure_reason() << " \"" + par_input + "\"\n";
            std::cerr << "Aborting...\n";
            stbi_image_free(alt_img);
            exit(1);
        }

        std::cout << "Comparing " << alt_input << " and " << par_input << std::endl;

        /* Make sure Local and Alternate outputs are the same */
        int err_cnt = 0;
        for(int y = 0; y < height; ++y) {
            for(int x = 0; x < width; ++x) {
                if(par_img[x + y * width] != alt_img[x + y * width]) {
                    ++err_cnt;
                }
            }
        }
        if(err_cnt == 0)
            std::cout << "CUDA and Sequential images are identical\n";
        else
            std::cout << err_cnt << " pixels are mismatched\n";

        /* Let go of STB image buffers */
        stbi_image_free(alt_img);
        stbi_image_free(par_img);
    }

    return 0;
}
